
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
using namespace std;

int main()
{
    // video frame parameters
    int WIDTH = 352;
    int HEIGHT = 288;
    size_t frame_bytes = WIDTH*HEIGHT*sizeof(uint8_t);

    // allocation frame buffers
    uint8_t* ref_frame; uint8_t* cur_frame;
    ref_frame = (uint8_t*) malloc(frame_bytes);
    cur_frame = (uint8_t*) malloc(frame_bytes);
    if(ref_frame == NULL) {printf("Error: buffer allocation failed"); exit(0);}

    // open yuv file
    FILE* yuv_file;
    yuv_file = fopen("CIF.yuv", "rb");
    if(yuv_file==NULL) {printf("Error: NULL file \n");exit(0);}
    rewind(yuv_file);

    // read first frame
    size_t result;
    result = fread(ref_frame, 1, frame_bytes, yuv_file);
    if (result != frame_bytes) {printf("Error: file read to buffer failed"); exit (0);}
    fseek(yuv_file, frame_bytes/2, SEEK_CUR); // skip U, V components

    // read next frame
    result = fread(cur_frame, 1, frame_bytes, yuv_file);
    if (result != frame_bytes) {printf("Error: file read to buffer failed"); exit (0);}
    fseek(yuv_file, frame_bytes/2, SEEK_CUR);

    // print elements
    for(int i=0; i<50; i++)
    {
        printf("%hhu ", ref_frame[i]);
    }
    printf("\n");

    FILE* write_file;
    write_file = fopen("y_frames/test_frame", "wb");
    fwrite(ref_frame, 1, frame_bytes, write_file);
    

    free(ref_frame);
    free(cur_frame);
    fclose(yuv_file);
    return 0;
}