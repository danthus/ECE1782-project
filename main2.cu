
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>

#define BLK_SIZE 8

/*You can use the following for any CUDA function that returns cudaError_t type*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code == hipSuccess) return;

    fprintf(stderr,"Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
}

__global__ void block_matching(uint8_t* ref_frame, uint8_t* curr_frame, int* mv, int width, int height, int blk_size, int srch_range)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int offset_x = threadIdx.x - srch_range;
    int offset_y = threadIdx.y - srch_range;
    int curr_idx = iy*width + ix;
    int ref_idx = curr_idx + offset_y*width + offset_x;
    int SAD = 0;
    if(ref_idx < 0 || ref_idx > width*height - (BLK_SIZE*width+BLK_SIZE))
    {
        ref_idx = curr_idx; 
        SAD = 999999; // give large value for outsiders
    }

    __shared__ int SAD_list[100];
    __shared__ int mv_list[200];

    for(int i=0; i<BLK_SIZE; i++)
    {
        for(int j=0; j<BLK_SIZE; j++)
        {
            SAD += abs((int)ref_frame[ref_idx+i*width+j] - (int)curr_frame[curr_idx+i*width+j]);
        }
    }

    SAD_list[threadIdx.y*2*srch_range + threadIdx.x] = SAD;
    mv_list[threadIdx.y*2*srch_range*2 + threadIdx.x*2] = offset_y;
    mv_list[threadIdx.y*2*srch_range*2 + threadIdx.x*2+1] = offset_x;
    __syncthreads();

    if(threadIdx.x == 0 && threadIdx.y == 0)
    {
        int min_SAD = SAD_list[0];
        int best_y = 0;
        int best_x = 0; 
        for(int k=1; k<2*srch_range*2*srch_range; k++)
        {
            if(min_SAD >= SAD_list[k])
            {
                min_SAD = SAD_list[k];
                best_y = mv_list[k];
                best_x = mv_list[k+1];
            }
        }
        mv[blockIdx.y*gridDim.x*4 + blockIdx.x*4] = blockIdx.y * blockDim.y;
        mv[blockIdx.y*gridDim.x*4 + blockIdx.x*4 + 1] = blockIdx.x * blockDim.x;
        mv[blockIdx.y*gridDim.x*4 + blockIdx.x*4 + 2] = best_y;
        mv[blockIdx.y*gridDim.x*4 + blockIdx.x*4 + 3] = best_x;
    }

}

bool read_next_frame(FILE* yuv_file, uint8_t* frame_buffer, size_t frame_bytes)
{
    size_t result;
    result = fread(frame_buffer, 1, frame_bytes, yuv_file);
    fseek(yuv_file, frame_bytes/2, SEEK_CUR); // skip U, V components
    if (result == frame_bytes) 
        return 1;
    else
        return 0;
}

// usage: main.cu <video file> <WIDTH> <HEIGHT> <BLK_size> <search range>
int main( int argc, char *argv[])
{
    // if( argc != 3) {
    //     printf( "Error: wrong number of args\n");
    //     exit(1);
    // }
    
    int WIDTH = atoi(argv[1]);
    int HEIGHT = atoi(argv[2]);
    // int BLK_SIZE = atoi(argv[3]);
    int SRC_range = atoi(argv[3]);

    // video file preprocess

    // !!!no padding right now, need to handle it!!!
    size_t pixels = WIDTH*HEIGHT;
    size_t frame_bytes = pixels*sizeof(uint8_t);
    size_t vector_size = 4*(pixels/BLK_SIZE/BLK_SIZE); // format: y x mv_y mv_x ...
    size_t vector_bytes = vector_size*sizeof(int);

    // process frame
    FILE* yuv_file;
    yuv_file = fopen("CIF.yuv", "rb");
    if(yuv_file==NULL) {printf("Error: NULL file \n");exit(0);}
    rewind(yuv_file);

    // Host memory allocation
    uint8_t *h_ref_frame, *h_cur_frame;
    int *h_mv;
    gpuErrchk(hipHostMalloc((void **)&h_ref_frame, frame_bytes, hipHostMallocDefault));
    gpuErrchk(hipHostMalloc((void **)&h_cur_frame, frame_bytes, hipHostMallocDefault));
    gpuErrchk(hipHostMalloc((void **)&h_mv, vector_bytes, hipHostMallocDefault));

    int ret = read_next_frame(yuv_file, h_ref_frame, frame_bytes);
    printf("ret = %d \n", ret);
    read_next_frame(yuv_file, h_cur_frame, frame_bytes);

    for(int i=0; i < 20; i++)
        printf("%hhu ", h_cur_frame[i]);
    printf("\n");

    dim3 dimGrid((WIDTH+BLK_SIZE-1)/BLK_SIZE, (HEIGHT+BLK_SIZE-1)/BLK_SIZE);
    dim3 dimBlock(BLK_SIZE, BLK_SIZE);

    // Device memory allocation
    uint8_t *d_ref_frame, *d_cur_frame;
    int *d_mv;
    hipMalloc((void **)&d_ref_frame, frame_bytes);
    hipMalloc((void **)&d_cur_frame, frame_bytes);
    hipMalloc((void **)&d_mv, vector_bytes);

    hipMemcpy(d_ref_frame, h_ref_frame, frame_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_cur_frame, h_cur_frame, frame_bytes, hipMemcpyHostToDevice);

    block_matching<<<dimGrid, dimBlock>>>(d_ref_frame, d_cur_frame, d_mv, WIDTH, HEIGHT, BLK_SIZE, SRC_range);

    hipDeviceSynchronize();

    hipMemcpy(h_mv, d_mv, vector_bytes, hipMemcpyDeviceToHost);

    // do compression using h_mv

    for(int i=17*44*4; i<17*44*4+64; i++)
        printf("%d ", h_mv[i]);
    printf("\n");


    hipHostFree(h_ref_frame);
    hipHostFree(h_cur_frame);
    hipHostFree(h_mv);
    hipFree(d_ref_frame);
    hipFree(d_cur_frame);
    hipFree(d_mv);
    fclose(yuv_file);
    return 0;
}